//#include "stdafx.h"                                                  // ����Ԥ����ͷ�ļ�

#include "devMatchCuda.cuh"                                           // ���뵼����������ͷ�ļ�


bool initCUDA(void)                                                  //  CUDA��ʼ������
{
	int   count = 0;

	printf("Start to detecte devices.........\n");                   //  ��ʾ��⵽���豸��

	hipGetDeviceCount(&count);                                     //   �������������ڵ���1.0���豸��

	if (count == 0){

		fprintf(stderr, "There is no device.\n");

		return false;

	}


	printf("%d device/s detected.\n", count);                      //   ��ʾ��⵽���豸��


	int i;

	for (i = 0; i < count; i++){                                  //  ������֤��⵽���豸�Ƿ�֧��CUDA

		hipDeviceProp_t prop;

		if (hipGetDeviceProperties(&prop, i) == hipSuccess) { //  ����豸���Բ���֤�Ƿ���ȷ

			if (prop.major >= 1)                                //  ��֤�����������������������ĵ�һλ���Ƿ����1

			{
				printf("Device %d: %s supports CUDA %d.%d.\n", i + 1, prop.name, prop.major, prop.minor);//��ʾ��⵽���豸֧�ֵ�CUDA�汾
				break;


			}
		}
	}

	if (i == count) {                                         //   û��֧��CUDA1.x���豸
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);                                       //    �����豸Ϊ�����̵߳ĵ�ǰ�豸

	return true;

}








